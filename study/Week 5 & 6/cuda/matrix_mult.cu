
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <functional>
#include <iostream>
#include <vector>
#include <chrono>

using namespace std::chrono;
using std::cout;
using std::generate;
using std::vector;

__global__ void matrixMul(const int *a, const int *b, int *c, int N) {
  // Compute each thread's global row and column index
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  // Iterate over row, and down column
  c[row * N + col] = 0;
  for (int k = 0; k < N; k++) {
    // Accumulate results for a single element
    c[row * N + col] += a[row * N + k] * b[k * N + col];
  }
}

// Check result on the CPU
void verify_result(vector<int> &a, vector<int> &b, vector<int> &c, int N) {
  // For every row...
  for (int i = 0; i < N; i++) {
    // For every column...
    for (int j = 0; j < N; j++) {
      // For every element in the row-column pair
      int tmp = 0;
      for (int k = 0; k < N; k++) {
        // Accumulate the partial results
        tmp += a[i * N + k] * b[k * N + j];
      }

      // Check against the CPU result
      assert(tmp == c[i * N + j]);
    }
  }
}

int main() {
    
  auto main_start = high_resolution_clock::now();  
  int N = 1 << 13;

  // Size (in bytes) of matrix
  size_t bytes = N * N * sizeof(int);

  auto start = high_resolution_clock::now();
  // Host vectors
  vector<int> h_a(N * N);
  vector<int> h_b(N * N);
  vector<int> h_c(N * N);

  
  // Initialize matrices
  generate(h_a.begin(), h_a.end(), []() { return rand() % 100; });
  generate(h_b.begin(), h_b.end(), []() { return rand() % 100; });

  
  // Allocate device memory
  int *d_a, *d_b, *d_c;
  hipMalloc(&d_a, bytes);
  hipMalloc(&d_b, bytes);
  hipMalloc(&d_c, bytes);


  // Copy data to the device

  

  hipMemcpy(d_a, h_a.data(), bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b.data(), bytes, hipMemcpyHostToDevice);

  auto stop = high_resolution_clock::now();
  auto duration = duration_cast<microseconds>(stop - start);
  cout << "Data transfer time from CPU - GPU : " << duration.count() << "\n";

  
  
  // Threads per CTA dimension
  int THREADS = 32;

  // Blocks per grid dimension (assumes THREADS divides N evenly)
  int BLOCKS = N / THREADS;

  // Use dim3 structs for block  and grid dimensions
  dim3 threads(THREADS, THREADS);
  dim3 blocks(BLOCKS, BLOCKS);

  
  
  

  // Launch kernel

  start = high_resolution_clock::now();

  matrixMul<<<blocks, threads>>>(d_a, d_b, d_c, N);

  stop = high_resolution_clock::now();
  duration = duration_cast<microseconds>(stop - start);
  cout << "cuda kernel time from CPU - GPU : " << duration.count() << "\n";




  // Copy back to the host

  start = high_resolution_clock::now();

  hipMemcpy(h_c.data(), d_c, bytes, hipMemcpyDeviceToHost);

  stop = high_resolution_clock::now();
  duration = duration_cast<microseconds>(stop - start);
  cout << "Data transfer time from GPU - CPU : " << duration.count() << "\n";

  // Check result
  // verify_result(h_a, h_b, h_c, N);

  // Free memory on device
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  auto main_stop = high_resolution_clock::now(); 
  duration = duration_cast<microseconds>(main_stop - main_start); 
  cout << "Total time : " << duration.count() << "\n";

   cout << "COMPLETED SUCCESSFULLY\n";

  return 0;
}