#include <stdio.h>
#include <hip/hip_runtime.h>
__global__ void dkernel(){
    printf("Hi from thread id %d\n", threadIdx.x);
}
int main() {
    dkernel<<<1,32>>>();
    hipDeviceSynchronize();
    return 0;
}